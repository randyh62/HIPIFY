// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --amap --default-preprocessor --experimental --roc %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocrand/rocrand.h"
// CHECK-NEXT: #include "rocrand/rocrand_kernel.h"
#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"
// CHECK-NOT: #include "rocrand/rocrand.h"
// CHECK-NOT: #include "rocrand/rocrand_kernel.h"

int main() {
  printf("23. cuRAND API to rocRAND API synthetic test\n");

  unsigned int *outputPtr = nullptr;
  unsigned int *constants = nullptr;
  unsigned long long *constantsLL = nullptr;
  float *outputPtrFloat = nullptr;
  double *outputPtrDouble = nullptr;
  unsigned int num_dimensions = 0;
  unsigned long long *outputPtrUll = nullptr;
  unsigned long long offset = 0;
  int version = 0;
  size_t num = 0;
  float mean = 0.f;
  double dmean = 0.f;
  float stddev = 0.f;
  double dstddev = 0.f;
  double dlambda = 0.f;

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  // CHECK: rocrand_status randStatus;
  // CHECK-NEXT: rocrand_status status;
  // CHECK-NEXT: rocrand_status STATUS_SUCCESS = ROCRAND_STATUS_SUCCESS;
  // CHECK-NEXT: rocrand_status STATUS_VERSION_MISMATCH = ROCRAND_STATUS_VERSION_MISMATCH;
  // CHECK-NEXT: rocrand_status STATUS_NOT_INITIALIZED = ROCRAND_STATUS_NOT_CREATED;
  // CHECK-NEXT: rocrand_status STATUS_ALLOCATION_FAILED = ROCRAND_STATUS_ALLOCATION_FAILED;
  // CHECK-NEXT: rocrand_status STATUS_TYPE_ERROR = ROCRAND_STATUS_TYPE_ERROR;
  // CHECK-NEXT: rocrand_status STATUS_OUT_OF_RANGE = ROCRAND_STATUS_OUT_OF_RANGE;
  // CHECK-NEXT: rocrand_status STATUS_LENGTH_NOT_MULTIPLE = ROCRAND_STATUS_LENGTH_NOT_MULTIPLE;
  // CHECK-NEXT: rocrand_status STATUS_DOUBLE_PRECISION_REQUIRED = ROCRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  // CHECK-NEXT: rocrand_status STATUS_LAUNCH_FAILURE = ROCRAND_STATUS_LAUNCH_FAILURE;
  // CHECK-NEXT: rocrand_status STATUS_INTERNAL_ERROR = ROCRAND_STATUS_INTERNAL_ERROR;
  hiprandStatus randStatus;
  hiprandStatus_t status;
  hiprandStatus_t STATUS_SUCCESS = HIPRAND_STATUS_SUCCESS;
  hiprandStatus_t STATUS_VERSION_MISMATCH = HIPRAND_STATUS_VERSION_MISMATCH;
  hiprandStatus_t STATUS_NOT_INITIALIZED = HIPRAND_STATUS_NOT_INITIALIZED;
  hiprandStatus_t STATUS_ALLOCATION_FAILED = HIPRAND_STATUS_ALLOCATION_FAILED;
  hiprandStatus_t STATUS_TYPE_ERROR = HIPRAND_STATUS_TYPE_ERROR;
  hiprandStatus_t STATUS_OUT_OF_RANGE = HIPRAND_STATUS_OUT_OF_RANGE;
  hiprandStatus_t STATUS_LENGTH_NOT_MULTIPLE = HIPRAND_STATUS_LENGTH_NOT_MULTIPLE;
  hiprandStatus_t STATUS_DOUBLE_PRECISION_REQUIRED = HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  hiprandStatus_t STATUS_LAUNCH_FAILURE = HIPRAND_STATUS_LAUNCH_FAILURE;
  hiprandStatus_t STATUS_INTERNAL_ERROR = HIPRAND_STATUS_INTERNAL_ERROR;

  // CHECK: rocrand_rng_type randRngType;
  // CHECK-NEXT: rocrand_rng_type randRngType_t;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_DEFAULT = ROCRAND_RNG_PSEUDO_DEFAULT;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_XORWOW = ROCRAND_RNG_PSEUDO_XORWOW;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_MRG32K3A = ROCRAND_RNG_PSEUDO_MRG32K3A;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_MTGP32 = ROCRAND_RNG_PSEUDO_MTGP32;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_MT19937 = ROCRAND_RNG_PSEUDO_MT19937;
  // CHECK-NEXT: rocrand_rng_type RNG_PSEUDO_PHILOX4_32_10 = ROCRAND_RNG_PSEUDO_PHILOX4_32_10;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_DEFAULT = ROCRAND_RNG_QUASI_DEFAULT;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SOBOL32 = ROCRAND_RNG_QUASI_SOBOL32;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SCRAMBLED_SOBOL32 = ROCRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SOBOL64 = ROCRAND_RNG_QUASI_SOBOL64;
  // CHECK-NEXT: rocrand_rng_type RNG_QUASI_SCRAMBLED_SOBOL64 = ROCRAND_RNG_QUASI_SCRAMBLED_SOBOL64;
  hiprandRngType_t randRngType;
  hiprandRngType_t randRngType_t;
  hiprandRngType_t RNG_PSEUDO_DEFAULT = HIPRAND_RNG_PSEUDO_DEFAULT;
  hiprandRngType_t RNG_PSEUDO_XORWOW = HIPRAND_RNG_PSEUDO_XORWOW;
  hiprandRngType_t RNG_PSEUDO_MRG32K3A = HIPRAND_RNG_PSEUDO_MRG32K3A;
  hiprandRngType_t RNG_PSEUDO_MTGP32 = HIPRAND_RNG_PSEUDO_MTGP32;
  hiprandRngType_t RNG_PSEUDO_MT19937 = HIPRAND_RNG_PSEUDO_MT19937;
  hiprandRngType_t RNG_PSEUDO_PHILOX4_32_10 = HIPRAND_RNG_PSEUDO_PHILOX4_32_10;
  hiprandRngType_t RNG_QUASI_DEFAULT = HIPRAND_RNG_QUASI_DEFAULT;
  hiprandRngType_t RNG_QUASI_SOBOL32 = HIPRAND_RNG_QUASI_SOBOL32;
  hiprandRngType_t RNG_QUASI_SCRAMBLED_SOBOL32 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
  hiprandRngType_t RNG_QUASI_SOBOL64 = HIPRAND_RNG_QUASI_SOBOL64;
  hiprandRngType_t RNG_QUASI_SCRAMBLED_SOBOL64 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64;

  // CHECK: rocrand_ordering randOrdering;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_PSEUDO_BEST = ROCRAND_ORDERING_PSEUDO_BEST;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_PSEUDO_DEFAULT = ROCRAND_ORDERING_PSEUDO_DEFAULT;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_PSEUDO_SEEDED = ROCRAND_ORDERING_PSEUDO_SEEDED;
  // CHECK-NEXT: rocrand_ordering RAND_ORDERING_QUASI_DEFAULT = ROCRAND_ORDERING_QUASI_DEFAULT;
  hiprandOrdering randOrdering;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_BEST = HIPRAND_ORDERING_PSEUDO_BEST;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_DEFAULT = HIPRAND_ORDERING_PSEUDO_DEFAULT;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_SEEDED = HIPRAND_ORDERING_PSEUDO_SEEDED;
  hiprandOrdering_t RAND_ORDERING_QUASI_DEFAULT = HIPRAND_ORDERING_QUASI_DEFAULT;

  // CHECK: rocrand_direction_vector_set directionVectorSet;
  // CHECK-NEXT: rocrand_direction_vector_set directionVectorSet_t;
  // CHECK-NEXT: rocrand_direction_vector_set DIRECTION_VECTORS_32_JOEKUO6 = ROCRAND_DIRECTION_VECTORS_32_JOEKUO6;
  // CHECK-NEXT: rocrand_direction_vector_set SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6 = ROCRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6;
  // CHECK-NEXT: rocrand_direction_vector_set DIRECTION_VECTORS_64_JOEKUO6 = ROCRAND_DIRECTION_VECTORS_64_JOEKUO6;
  // CHECK-NEXT: rocrand_direction_vector_set SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6 = ROCRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6;
  hiprandDirectionVectorSet_t directionVectorSet;
  hiprandDirectionVectorSet_t directionVectorSet_t;
  hiprandDirectionVectorSet_t DIRECTION_VECTORS_32_JOEKUO6 = HIPRAND_DIRECTION_VECTORS_32_JOEKUO6;
  hiprandDirectionVectorSet_t SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6 = HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6;
  hiprandDirectionVectorSet_t DIRECTION_VECTORS_64_JOEKUO6 = HIPRAND_DIRECTION_VECTORS_64_JOEKUO6;
  hiprandDirectionVectorSet_t SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6 = HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6;

  // CHECK: rocrand_generator_base_type *randGenerator_st = nullptr;
  // CHECK-NEXT: rocrand_generator randGenerator;
  hiprandGenerator_st *randGenerator_st = nullptr;
  hiprandGenerator_t randGenerator;

  // CHECK: rocrand_device::sobol64_engine<false> randStateSobol64;
  // CHECK-NEXT: rocrand_state_sobol64 randStateSobol64_t;
  hiprandStateSobol64 randStateSobol64;
  hiprandStateSobol64_t randStateSobol64_t;

  // CHECK: rocrand_device::scrambled_sobol64_engine<false> randStateScrambledSobol64;
  // CHECK-NEXT: rocrand_state_scrambled_sobol64 randStateScrambledSobol64_t;
  hiprandStateScrambledSobol64 randStateScrambledSobol64;
  hiprandStateScrambledSobol64_t randStateScrambledSobol64_t;

  // CHECK: rocrand_device::sobol32_engine<false> randStateSobol32;
  // CHECK-NEXT: rocrand_state_sobol32 randStateSobol32_t;
  hiprandStateSobol32 randStateSobol32;
  hiprandStateSobol32_t randStateSobol32_t;

  // CHECK: rocrand_device::scrambled_sobol32_engine<false> randStateScrambledSobol32;
  // CHECK-NEXT: rocrand_state_scrambled_sobol32 randStateScrambledSobol32_t;
  hiprandStateScrambledSobol32 randStateScrambledSobol32;
  hiprandStateScrambledSobol32_t randStateScrambledSobol32_t;

  // CHECK: rocrand_discrete_distribution_st *discreteDistribution_st = nullptr;
  // CHECK-NEXT: rocrand_discrete_distribution discreteDistribution_t = nullptr;
  hiprandDiscreteDistribution_st *discreteDistribution_st = nullptr;
  hiprandDiscreteDistribution_t discreteDistribution_t = nullptr;

  // CHECK: rocrand_device::mtgp32_engine stateMtgp32;
  // CHECK-NEXT: rocrand_state_mtgp32 stateMtgp32_t;
  hiprandStateMtgp32 stateMtgp32;
  hiprandStateMtgp32_t stateMtgp32_t;

  // CHECK: rocrand_device::mrg32k3a_engine stateMRG32k3a;
  // CHECK-NEXT: rocrand_state_mrg32k3a stateMRG32k3a_t;
  hiprandStateMRG32k3a stateMRG32k3a;
  hiprandStateMRG32k3a_t stateMRG32k3a_t;

  // CHECK: rocrand_device::philox4x32_10_engine statePhilox4_32_10;
  // CHECK-NEXT: rocrand_state_philox4x32_10 statePhilox4_32_10_t;
  hiprandStatePhilox4_32_10 statePhilox4_32_10;
  hiprandStatePhilox4_32_10_t statePhilox4_32_10_t;

  // CUDA: curandStatus_t CURANDAPI curandCreateGenerator(curandGenerator_t *generator, curandRngType_t rng_type);
  // ROC: rocrand_status ROCRANDAPI rocrand_create_generator(rocrand_generator * generator, rocrand_rng_type rng_type);
  // CHECK: status = rocrand_create_generator(&randGenerator, randRngType_t);
  status = hiprandCreateGenerator(&randGenerator, randRngType_t);

  // CUDA: curandStatus_t CURANDAPI curandDestroyGenerator(curandGenerator_t generator);
  // ROC: rocrand_status ROCRANDAPI rocrand_destroy_generator(rocrand_generator generator);
  // CHECK: status = rocrand_destroy_generator(randGenerator);
  status = hiprandDestroyGenerator(randGenerator);

  // CUDA: curandStatus_t CURANDAPI curandCreateGeneratorHost(curandGenerator_t *generator, curandRngType_t rng_type);
  // ROC: rocrand_status ROCRANDAPI rocrand_create_generator_host_blocking(rocrand_generator* generator, rocrand_rng_type rng_type);
  // CHECK: status = rocrand_create_generator_host_blocking(&randGenerator, randRngType_t);
  status = hiprandCreateGeneratorHost(&randGenerator, randRngType_t);

  // CUDA: curandStatus_t CURANDAPI curandGenerate(curandGenerator_t generator, unsigned int *outputPtr, size_t num);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate(rocrand_generator generator, unsigned int * output_data, size_t n);
  // CHECK: status = rocrand_generate(randGenerator, outputPtr, num);
  status = hiprandGenerate(randGenerator, outputPtr, num);

  // CUDA: curandStatus_t CURANDAPI curandGenerateLogNormal(curandGenerator_t generator, float *outputPtr, size_t n, float mean, float stddev);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_log_normal(rocrand_generator generator, float * output_data, size_t n, float mean, float stddev);
  // CHECK: status = rocrand_generate_log_normal(randGenerator, outputPtrFloat, num, mean, stddev);
  status = hiprandGenerateLogNormal(randGenerator, outputPtrFloat, num, mean, stddev);

  // CUDA: curandStatus_t CURANDAPI curandGenerateLongLong(curandGenerator_t generator, unsigned long long *outputPtr, size_t num);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_long_long(rocrand_generator generator, unsigned long long int* output_data, size_t n);
  // CHECK: status = rocrand_generate_long_long(randGenerator, outputPtrUll, num);
  status = hiprandGenerateLongLong(randGenerator, outputPtrUll, num);

  // CUDA: curandStatus_t CURANDAPI curandGenerateNormal(curandGenerator_t generator, float *outputPtr, size_t n, float mean, float stddev);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_normal(rocrand_generator generator, float * output_data, size_t n, float mean, float stddev);
  // CHECK: status = rocrand_generate_normal(randGenerator, outputPtrFloat, num, mean, stddev);
  status = hiprandGenerateNormal(randGenerator, outputPtrFloat, num, mean, stddev);

  // CUDA: curandStatus_t CURANDAPI curandGenerateNormalDouble(curandGenerator_t generator, double *outputPtr, size_t n, double mean, double stddev);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_normal_double(rocrand_generator generator, double * output_data, size_t n, double mean, double stddev);
  // CHECK: status = rocrand_generate_normal_double(randGenerator, outputPtrDouble, num, dmean, dstddev);
  status = hiprandGenerateNormalDouble(randGenerator, outputPtrDouble, num, dmean, dstddev);

  // CUDA: curandStatus_t CURANDAPI curandGenerateUniform(curandGenerator_t generator, float *outputPtr, size_t num);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_uniform(rocrand_generator generator, float * output_data, size_t n);
  // CHECK: status = rocrand_generate_uniform(randGenerator, outputPtrFloat, num);
  status = hiprandGenerateUniform(randGenerator, outputPtrFloat, num);

  // CUDA: curandStatus_t CURANDAPI curandGenerateUniformDouble(curandGenerator_t generator, double *outputPtr, size_t num);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_uniform_double(rocrand_generator generator, double * output_data, size_t n);
  // CHECK: status = rocrand_generate_uniform_double(randGenerator, outputPtrDouble, num);
  status = hiprandGenerateUniformDouble(randGenerator, outputPtrDouble, num);

  // CUDA: curandStatus_t CURANDAPI curandGenerateLogNormalDouble(curandGenerator_t generator, double *outputPtr, size_t n, double mean, double stddev);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_log_normal_double(rocrand_generator generator, double * output_data, size_t n, double mean, double stddev);
  // CHECK: status = rocrand_generate_log_normal_double(randGenerator, outputPtrDouble, num, dmean, dstddev);
  status = hiprandGenerateLogNormalDouble(randGenerator, outputPtrDouble, num, dmean, dstddev);

  // CUDA: curandStatus_t CURANDAPI curandGeneratePoisson(curandGenerator_t generator, unsigned int *outputPtr, size_t n, double lambda);
  // ROC: rocrand_status ROCRANDAPI rocrand_generate_poisson(rocrand_generator generator, unsigned int * output_data, size_t n, double lambda);
  // CHECK: status = rocrand_generate_poisson(randGenerator, outputPtr, num, dlambda);
  status = hiprandGeneratePoisson(randGenerator, outputPtr, num, dlambda);

  // CUDA: curandStatus_t CURANDAPI curandGenerateSeeds(curandGenerator_t generator);
  // ROC: rocrand_status ROCRANDAPI rocrand_initialize_generator(rocrand_generator generator);
  // CHECK: status = rocrand_initialize_generator(randGenerator);
  status = hiprandGenerateSeeds(randGenerator);

  // CUDA: curandStatus_t CURANDAPI curandSetGeneratorOffset(curandGenerator_t generator, unsigned long long offset);
  // ROC: rocrand_status ROCRANDAPI rocrand_set_offset(rocrand_generator generator, unsigned long long offset);
  // CHECK: status = rocrand_set_offset(randGenerator, offset);
  status = hiprandSetGeneratorOffset(randGenerator, offset);

  // CUDA: curandStatus_t CURANDAPI curandSetPseudoRandomGeneratorSeed(curandGenerator_t generator, unsigned long long seed);
  // ROC: rocrand_status ROCRANDAPI rocrand_set_seed(rocrand_generator generator, unsigned long long seed);
  // CHECK: status = rocrand_set_seed(randGenerator, offset);
  status = hiprandSetPseudoRandomGeneratorSeed(randGenerator, offset);

  // CUDA: curandStatus_t CURANDAPI curandSetStream(curandGenerator_t generator, cudaStream_t stream);
  // ROC: rocrand_status ROCRANDAPI rocrand_set_stream(rocrand_generator generator, hipStream_t stream);
  // CHECK: status = rocrand_set_stream(randGenerator, stream);
  status = hiprandSetStream(randGenerator, stream);

  // CUDA: curandStatus_t CURANDAPI curandCreatePoissonDistribution(double lambda, curandDiscreteDistribution_t *discrete_distribution);
  // ROC: rocrand_status ROCRANDAPI rocrand_create_poisson_distribution(double lambda, rocrand_discrete_distribution * discrete_distribution);
  // CHECK: status = rocrand_create_poisson_distribution(dlambda, &discreteDistribution_t);
  status = hiprandCreatePoissonDistribution(dlambda, &discreteDistribution_t);

  // CUDA: curandStatus_t CURANDAPI curandDestroyDistribution(curandDiscreteDistribution_t discrete_distribution);
  // ROC: rocrand_status ROCRANDAPI rocrand_destroy_discrete_distribution(rocrand_discrete_distribution discrete_distribution);
  // CHECK: status = rocrand_destroy_discrete_distribution(discreteDistribution_t);
  status = hiprandDestroyDistribution(discreteDistribution_t);

  // CUDA: curandStatus_t CURANDAPI curandGetScrambleConstants32(unsigned int * * constants);
  // ROC: rocrand_status ROCRANDAPI rocrand_get_scramble_constants32(const unsigned int** constants);
  // CHECK: status = rocrand_get_scramble_constants32(&constants);
  status = hiprandGetScrambleConstants32(&constants);

  // CUDA: curandStatus_t CURANDAPI curandGetScrambleConstants64(unsigned long long * * constants);
  // ROC: rocrand_status ROCRANDAPI rocrand_get_scramble_constants64(const unsigned long long** constants);
  // CHECK: status = rocrand_get_scramble_constants64(&constantsLL);
  status = hiprandGetScrambleConstants64(&constantsLL);

  // CUDA: curandStatus_t CURANDAPI curandGetVersion(int *version);
  // ROC: rocrand_status ROCRANDAPI rocrand_get_version(int * version);
  // CHECK: status = rocrand_get_version(&version);
  status = hiprandGetVersion(&version);

  // CUDA: curandStatus_t CURANDAPI curandSetGeneratorOrdering(curandGenerator_t generator, curandOrdering_t order);
  // ROC: rocrand_status ROCRANDAPI rocrand_set_ordering(rocrand_generator generator, rocrand_ordering order);
  // CHECK: status = rocrand_set_ordering(randGenerator, randOrdering);
  status = hiprandSetGeneratorOrdering(randGenerator, randOrdering);

  // CUDA: curandStatus_t CURANDAPI curandSetQuasiRandomGeneratorDimensions(curandGenerator_t generator, unsigned int num_dimensions);
  // ROC: rocrand_status ROCRANDAPI rocrand_set_quasi_random_generator_dimensions(rocrand_generator generator, unsigned int dimensions);
  // CHECK: status = rocrand_set_quasi_random_generator_dimensions(randGenerator, num_dimensions);
  status = hiprandSetQuasiRandomGeneratorDimensions(randGenerator, num_dimensions);

#if CUDA_VERSION >= 11000 && CURAND_VERSION >= 10200
  // CHECK: rocrand_ordering RAND_ORDERING_PSEUDO_LEGACY = ROCRAND_ORDERING_PSEUDO_LEGACY;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_LEGACY = HIPRAND_ORDERING_PSEUDO_LEGACY;
#endif

#if CUDA_VERSION >= 11050 && CURAND_VERSION >= 10207
  // CHECK: rocrand_ordering RAND_ORDERING_PSEUDO_DYNAMIC = ROCRAND_ORDERING_PSEUDO_DYNAMIC;
  hiprandOrdering_t RAND_ORDERING_PSEUDO_DYNAMIC = HIPRAND_ORDERING_PSEUDO_DYNAMIC;
#endif

  return 0;
}
