// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18.11030.12000. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: rocsparse_status status_t;
  hipsparseStatus_t status_t;

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_B, matDescr_C, matDescr_D;

  // CHECK: rocsparse_operation opA, opB, opX;
  hipsparseOperation_t opA, opB, opX;

  // CHECK: rocsparse_solve_policy solvePolicy_t;
  hipsparseSolvePolicy_t solvePolicy_t;

  int m = 0;
  int n = 0;
  int k = 0;
  int innz = 0;
  int nnza = 0;
  int nnzb = 0;
  int nnzc = 0;
  int nnzd = 0;
  int csrRowPtrA = 0;
  int csrRowPtrB = 0;
  int csrRowPtrC = 0;
  int csrRowPtrD = 0;
  int csrColIndA = 0;
  int csrColIndB = 0;
  int csrColIndC = 0;
  int csrColIndD = 0;
  int bufferSizeInBytes = 0;
  size_t bufferSize = 0;
  double dA = 0.f;
  double dB = 0.f;
  double dAlpha = 0.f;
  double dF = 0.f;
  double dX = 0.f;
  double dcsrSortedValA = 0.f;
  double dcsrSortedValB = 0.f;
  double dcsrSortedValC = 0.f;
  double dcsrSortedValD = 0.f;
  float fAlpha = 0.f;
  float fA = 0.f;
  float fB = 0.f;
  float fF = 0.f;
  float fX = 0.f;
  float csrSortedValA = 0.f;
  float csrSortedValB = 0.f;
  float csrSortedValC = 0.f;
  float csrSortedValD = 0.f;
  void *alpha = nullptr;
  void *pBuffer = nullptr;
  void *tempBuffer = nullptr;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuDoubleComplex -> rocsparse_double_complex under a new option --sparse
  // CHECK: rocblas_double_complex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dComplexcsrSortedValD, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;
  hipDoubleComplex dcomplex, dcomplexA, dcomplexAlpha, dcomplexB, dcomplexBeta, dcomplexC, dcomplexF, dcomplexX, dcomplexY, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValB, dComplexcsrSortedValC, dComplexcsrSortedValD, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx, dcomplex_boost_val;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuComplex -> rocsparse_float_complex under a new option --sparse
  // CHECK: rocblas_float_complex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complexcsrSortedValD, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;
  hipComplex complex, complexA, complexAlpha, complexB, complexBeta, complexC, complexF, complexX, complexY, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValB, complexcsrSortedValC, complexcsrSortedValD, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx, complex_boost_val;

#if CUDA_VERSION >= 8000
  // TODO: [#899] There should be rocsparse_datatype instead of hipDataType
  hipDataType dataType_t;
  hipDataType dataType;
#endif

#if (CUDA_VERSION >= 10010 && CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: rocsparse_spmat_descr spMatDescr_t, spmatA, spmatB, spmatC;
  hipsparseSpMatDescr_t spMatDescr_t, spmatA, spmatB, spmatC;

  // CHECK: rocsparse_dnmat_descr dnMatDescr_t, dnmatA, dnmatB, dnmatC;
  hipsparseDnMatDescr_t dnMatDescr_t, dnmatA, dnmatB, dnmatC;
#endif

#if CUDA_VERSION >= 11030 && CUSPARSE_VERSION >= 11600
  // CHECK: rocsparse_spsm_alg spSMAlg_t;
  hipsparseSpSMAlg_t spSMAlg_t;

  // NOTE:cusparseSpSMDescr_t doesn't have a correspondence in rocSPARSE, the corresponding function argument is removed in the hipified call of the rocsparse_spsm function
  hipsparseSpSMDescr_t spSMDescr;

#if CUDA_VERSION < 12000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpSM_analysis(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseSpMatDescr_t matA, cusparseDnMatDescr_t matB, cusparseDnMatDescr_t matC, cudaDataType computeType, cusparseSpSMAlg_t alg, cusparseSpSMDescr_t spsmDescr, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spsm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, const void* alpha, const rocsparse_spmat_descr matA, const rocsparse_dnmat_descr matB, const rocsparse_dnmat_descr matC, rocsparse_datatype compute_type, rocsparse_spsm_alg alg, rocsparse_spsm_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spsm(handle_t, opA, opB, alpha, spmatA, dnmatB, dnmatC, dataType, spSMAlg_t, rocsparse_spsm_stage_compute, nullptr, tempBuffer);
  status_t = cusparseSpSM_analysis(handle_t, opA, opB, alpha, spmatA, dnmatB, dnmatC, dataType, spSMAlg_t, spSMDescr, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpSM_solve(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseSpMatDescr_t matA, cusparseDnMatDescr_t matB, cusparseDnMatDescr_t matC, cudaDataType computeType, cusparseSpSMAlg_t alg, cusparseSpSMDescr_t spsmDescr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spsm(rocsparse_handle handle, rocsparse_operation trans_A, rocsparse_operation trans_B, const void* alpha, const rocsparse_spmat_descr matA, const rocsparse_dnmat_descr matB, const rocsparse_dnmat_descr matC, rocsparse_datatype compute_type, rocsparse_spsm_alg alg, rocsparse_spsm_stage stage, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spsm(handle_t, opA, opB, alpha, spmatA, dnmatB, dnmatC, dataType, spSMAlg_t, rocsparse_spsm_stage_compute, nullptr, nullptr);
  status_t = cusparseSpSM_solve(handle_t, opA, opB, alpha, spmatA, dnmatB, dnmatC, dataType, spSMAlg_t, spSMDescr);
#endif
#endif

  return 0;
}
