
#include <hip/hip_runtime.h>
// RUN: %run_test hipify "%s" "%t" %hipify_args %clang_args

// CHECK: #include <hip/hip_runtime.h>

template <typename T>
void check(T result, char const *const func, const char *const file, int const line) {
  if (result) {
    fprintf(stderr, "Error at %s:%d code=%d(%s) \" \n", file, line, static_cast<unsigned int>(result), func);
    exit(EXIT_FAILURE);
  }
}

#define checkErrors(val) check((val), #val, __FILE__, __LINE__)
#define num 1024

int main(int argc, const char *argv[]) {
  int *input = nullptr;
  int deviceCount = 0;
  // CHECK: checkErrors(hipGetDeviceCount(&deviceCount));
  checkErrors(hipGetDeviceCount(&deviceCount));
  printf("Device Count: %d\n", deviceCount);
  // CHECK: hipDeviceProp_t deviceProp;
  hipDeviceProp_t deviceProp;
  deviceProp.major = 0;
  deviceProp.minor = 0;
  int deviceID = 0;
  // CHECK: checkErrors(hipGetDeviceProperties(&deviceProp, deviceID));
  checkErrors(hipGetDeviceProperties(&deviceProp, deviceID));
  // CHECK: checkErrors(hipSetDevice(deviceID));
  checkErrors(hipSetDevice(deviceID));
  // CHECK: checkErrors(hipHostMalloc(&input, sizeof(int) * num * 2, hipHostMallocDefault));
  checkErrors(hipHostMalloc(&input, sizeof(int) * num * 2, hipHostMallocDefault));
  for (int i = 0; i < num * 2; ++i) {
    input[i] = i;
  }
  // CHECK: checkErrors(hipHostFree(input));
  checkErrors(hipHostFree(input));
  // CHECK: checkErrors(hipDeviceSynchronize());
  checkErrors(hipDeviceSynchronize());
  return 0;
}
